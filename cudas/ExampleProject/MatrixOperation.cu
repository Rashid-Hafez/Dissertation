#include "hip/hip_runtime.h"
/**-------------------------------------------------------------------------------
Name:

@ Description:
- Program to multiply 2 matrices together.
-------------------------------------------------------------------------------**/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "MatrixOperation.h"
#include <stdio.h>
#define BLOCKSIZE 32
#define MAX(a,b) (a>b ? a:b)
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

int STRIDE;
int unified =0;
int PINNED = 0;
int nStreams;
long long GRIDSIZE = (long long)65535L*65535L; //Larger than this does not fit on GPU
int partitions = 2;
int BlockSIZE;
dim3 GRID; dim3 BLOCK;
static hipDeviceProp_t PROPS;
//-------------------------------------------------------------------------------

/****************************************** 

@Description: MULT_KERNEL

Row based square matrix multiplication with optimised shared memory kernel.

@Parameters:
	- A, B: 1 Dimensional row based arrays
	- C: result matrix
	- N: Size of row/column

********************************************/
__global__ void multiplication(int *A, int* B, int *C, int N, int BlockSIZE){
   int ROW = blockIdx.y*BlockSIZE+threadIdx.y; // BlockIndex * BlocksizeY + ThreadY
   int COL = blockIdx.x*BlockSIZE+threadIdx.x;
   int sum = 0;
   if (ROW < N && COL < N){ //TODO: ZERO OUT OF BOUNDS

   	for (int i = 0; i < N; ++i)
   	{
   		sum += A[ROW*N+i] * B[i * N + COL]; 
   	}
   	C[ROW*N+COL] = sum;
   }
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void MatrixOperation(int* aC, int* bC, int* cC, long long width1, long long height1, long long width2,
	long long height2, int **a,int** b,int** c, hipDeviceProp_t *prop){

	if(width1 != height2){
		printf("Error, width of first mat must = height of second mat\n");
		exit(0);
	}

	/*------------Basic Generic Setup------------------- */
	hipStream_t stream0;
	gpuErrchk(hipStreamCreate(&stream0));
	hipEvent_t start,stop;
	float time;
	/*----------------------------------------------------- */

	if (PINNED) //if page locked memory on matrix
	{	printf("Pinned... Partitioning\n");

	/*------------------Partition------------------------*/
		unsigned long long SplitA_Row;
		unsigned long long SplitB_Col;
		unsigned long long N;
		unsigned long long MaxData = height1 * width2; //total entries of A and B
		unsigned long long SubMatSize;

		SplitA_Row = ceil (height1/partitions) ;
		SplitB_Col = ceil (width2/partitions) ;
		//Make N at Least Half the grid size 
		N = SplitA_Row*SplitB_Col;

		while(N>=GRIDSIZE){
			if (N<GRIDSIZE) //Safety precaution
			{
				printf("N is solved\n");
				break;
			}
			if (N>= GRIDSIZE) //If our matrix is still too big then...
			{
				printf("N BIGGER THAN GRIDSIZE\n");
				SplitA_Row = ceil(SplitA_Row/partitions);
				SplitB_Col = ceil(SplitB_Col/partitions);
				N = SplitA_Row * SplitB_Col;
			}
		}
		
		SetupDim(SplitA_Row, SplitB_Col, *prop);
		
		gpuErrchk(hipEventCreate(&start));
		gpuErrchk(hipEventCreate(&stop));

		//Timer START LETS GOOO!
		gpuErrchk(hipEventRecord(start,0));
		//malloc
		printf("CudaMalloc\n");
		gpuErrchk(hipMalloc((void**)&aC, N));
		gpuErrchk(hipMalloc((void**)&bC, N));
		gpuErrchk(hipMalloc((void**)&cC, N));

		printf("a = \n");
/*---------------------ASYNC STREAM LOOP------------------------------*/
		for (int i = 0; i < MaxData; i+=N)
		{
			printf("%d\n",a+i ); 
			gpuErrchk(hipMemcpyAsync(aC,a,N*sizeof(int),hipMemcpyHostToDevice,stream0));
			gpuErrchk(hipMemcpyAsync(bC,b+i,N*sizeof(int),hipMemcpyHostToDevice,stream0));
			//									multiply									//
			multiplication<<<GRID,BLOCK,0,stream0>>>(aC,bC,cC,height1,BlockSIZE);
			//
			gpuErrchk(hipMemcpyAsync(*c+i,cC,N*sizeof(int),hipMemcpyDeviceToHost)); //i = N;
		}

		gpuErrchk(hipStreamSynchronize(stream0)); // Tell CPU to hold his horses and wait
		gpuErrchk(hipEventRecord(stop,0));
		gpuErrchk(hipEventSynchronize(stop));
		gpuErrchk(hipEventElapsedTime(&time, start, stop));

		printf("Time Taken: %3.1f ms/n \n",time);

		gpuErrchk(hipStreamDestroy(stream0));
	}
/*----------------------------------------------------------------------------*/

else if (unified)
{
	printf("unified\n");
}

/*--------------------- NO STREAM MULTIPLICATION------------------------------*/
	else{
		long size1 = width1 * height1 * sizeof(int); //matrixA
		long size2 = width2 * height2 * sizeof(int); //matrixB

		printf("Size1 = %d\n",size1);
		printf("Height1 = %d\n",height1);
		
		SetupDim(width1,height2,*prop);
		hipMalloc((void**)&aC, size1);
		hipMalloc((void**)&bC,size2);
		hipMalloc((void**)&cC, size2);

		gpuErrchk( hipPeekAtLastError() );
		//hipDeviceSynchronize();
		gpuErrchk(hipMemcpy(&c,cC,size1,hipMemcpyDeviceToHost));
	}
}


/**
**************************************************************************************************
Name: SetupDim

Description:
	Sets up entire grid dimensions. The amount of blocks to use to cover the grid depends on the matrix size.
**************************************************************************************************
**/
void SetupDim (long long width1, long long height2, hipDeviceProp_t prop){

	if (prop.major>=2)
	{
		int bblock = BLOCKSIZE;
		printf("Device compute is 2 or over, utilizing thread count\n");
		int gCol = ceil(width1/BLOCKSIZE);
		int gRow = ceil(height2/BLOCKSIZE);
		// printf("Grid is %d by %d \n",gCol,gRow );
		dim3 grid(gCol,gRow);
		dim3 block(bblock,bblock); //(BLOCKSIZE,BLOCKSIZE) 
		//32*32 threads per block. = 1024 Studies suggest this isn't always the most optimal.
		BlockSIZE = bblock;
		GRID = grid;
		BLOCK = block;
	}
	else{
		printf("Device Compute Capacity less than 2, reducing threadcount\n");
		BlockSIZE = 16;
		int gCol = ceil(width1/BLOCKSIZE);
		int gRow = ceil(height2/BLOCKSIZE);
		printf("Grid is %d by %d \n", gCol, gRow);
		dim3 grid(gCol,gRow);
		dim3 block(BlockSIZE,BlockSIZE); //(BLOCKSIZE,BLOCKSIZE); //32*32 threads per block. = 1024; Studies suggest this isn't always the most optimal.
		
		GRID = grid;
		BLOCK = block;
	}

	// if(gridDim.x*blockDim.x < size1){
	// 	STRIDE = 42; // we tell the kernel to use a stride method of multiplication.
	//	}
	//}
		//else if (compute >=3.x){}
		//else if (compute < 2.x){}
}

/**

Description:
Convert normal matrix to ROW MAJOR matrix, if the matrices are bigger than the GPU memory the function will use pinned memory (i.e. hostmalloc). 

a(i,j) can be flatten to 1D array b(k)
mat[0] to mat[m] = the first row, mat[m+1] = the second row. mat[2*m+1] = third row

@Param: 
  - mat : the 2D matrix to convert to 1D
  - n : amount of rows
  - m : amount of colombs in the matrix

  MOVE TO MAIN CLASS
**/
int * RowMajorMat(int** mat, long long n,long long m){
int * newMat;
unsigned long long ss = n*m;
 if(ss <= GRIDSIZE/10000){
 	if(!(newMat=SetupMat(ss))) return 0;
 } 
 else{
 	printf("Setting up mat for page locked storage\n");
 	if(!(newMat = SetupMat(ss))) return 0;
 }

  for (long i = 0; i<n; i++){
    for (long j =0; j<m; j++){
    long k = i * m + j;
      newMat[k] = mat[i][j];
    }
  }
  return newMat;
}

/**
Description:
Convert normal matrix to COLUMN MAJOR matrix, if the matrices are bigger than the GPU memory the function will use pinned memory (i.e. cudahostmalloc). 

a(i,j) can be flatten to 1D array b(k)
mat[0] to mat[m] = the first row, mat[m+1] = the second row. mat[2*m+1] = third row

@Param: 
  - mat : the 2D matrix to convert to 1D
  - n : amount of rows
  - m : amount of colombs in the matrix
**/
int * ColumnMajorMat(int** mat, long long n, long long m){
int * newMat;
unsigned long long ss = n*m;
 if(ss <= GRIDSIZE){
 	newMat = (int*) malloc(ss*sizeof(int));
 } 
 else{
 	printf("Setting up mat for page locked storage\n");
 	if(!(newMat = SetupMat(ss))) return 0;
 }

  for (long i = 0; i<m; i++){
    for (long j =0; j<n; j++){
    long k = i * n +j;
      newMat[k] = mat[j][i];
    }
  }
  return newMat;
}

//Setup for unified/pinned and others
int* SetupMat(long long size){

	if (!PROPS.canMapHostMemory)
	{
		printf("Pinned not supported\n");
		if(!PROPS.managedMemory){
		fprintf(stderr,"Unified AND Pinned memory not supported... exiting\n");
		return (0);
		}
	}

	int input;
	printf("Enter 1 for pinned, 2 for unified\n");
	scanf("%d",&input);
	if (input ==1) //PINNED MEM
	{ 
		printf("Pinned!\n");
		PINNED =42;
		int *mat;
		gpuErrchk(hipHostAlloc((void**)&mat,size*sizeof(int),hipHostMallocPortable)); //Page locked
		printf("Pinned success\n");
		return mat;
	}
	if (input == 2) //UNIFIED MEM
	{
		int*mat;
		/*
		unified = true; 
		hipMallocManaged() */
		return mat;
	}
	fprintf(stderr,"Something wrong?" );
return 0;
}

void setProp(int d){
	gpuErrchk(hipSetDevice(d));
	gpuErrchk(hipGetDeviceProperties(&PROPS,d));
}
hipDeviceProp_t getProp(){
	return(PROPS);
}