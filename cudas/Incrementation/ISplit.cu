#include "hip/hip_runtime.h"
/*
	Rashid Hafez

*/
#include "ISplit.h"

dim3 GRID; 
dim3 BLOCK;
static hipDeviceProp_t PROPS;

/******************************
  Increment Kernel
*******************************/
__global__ void Incr(float * aC, unsigned long n, unsigned long long it){

int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int offset = x + y * blockDim.x * gridDim.x; //works for any size and anything

  	if(offset<=n){
  	   	aC[offset]++;
  	 }
}


void ISplit(float * & arr, unsigned long sz, hipDeviceProp_t* prop){
	
}

void setProp(int d){
	gpuErrchk(hipSetDevice(d));
	gpuErrchk(hipGetDeviceProperties(&PROPS,d));
}
hipDeviceProp_t getProp(){
	return(PROPS);
}
