#include "hip/hip_runtime.h"
#include <time.h>       /* time */ 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include "ISplit.h" 
 
/////////////// MACROS and GLOBALS: ////////////// 
#define N 100000000 
#define oneGB 100000000 
#define BLOCK_SIZE 32 
unsigned long long gMem; int gSize[3]; int wSize; int TPB;//max threads per block 
///////////////////////////// 
 
typedef struct{ 
   
  unsigned long long M;//size 
  size_t p; //partition 
  unsigned int overflow;  
  float * vec; 
}VECTOR; 
 
////////////////////////////////////////////////////////// 
int CheckI(float *vv, unsigned long s, float *&c); 
 
void randomInit(float* &data) 
{ 
  #pragma unroll 
    for (int i = 0; i <= N; i++){ 
        data[i] = rand()% (1000 + 1 - 1) + 1; 
        } 
} 
////////////////////////////////////////////////////////// 
 
/////////////////////////////////////////////////////////////////////////////////////// 
int main(int argc, char** argv){ 

  //Setup Check// 
  int Dev = 0; 
  hipDeviceProp_t pp; 
  setProp(Dev); 
  pp = getProp(); 
   
  //timer 
  hipEvent_t start1,stop1;  
  float time1;
  gpuErrchk(hipEventCreate(&start1));  
  gpuErrchk(hipEventCreate(&stop1));  
  //Timer START LETS GOOO!  
  gpuErrchk(hipEventRecord(start1,0));  

  gMem = pp.totalGlobalMem; 
  gSize[0] = pp.maxGridSize[0]; gSize[1] = pp.maxGridSize[1]; gSize[2] = pp.maxGridSize[2]; 
  wSize = pp.warpSize; 
  TPB = pp.maxThreadsPerBlock; 
  //printf("total Global mem: %ld\n", gMem); 
  //printf("maxGridSize= %d,%d,%d \n",gSize[0],gSize[1],gSize[2]); 
  //printf("Warp Size: %d\n", wSize); 
  //printf(" TPB: %d\n", TPB); 
//----------------------------------------------------------- 
  srand(356); 

  printf("Initialised\n"); 
  //printf("Creating Template Matrix\n"); 

  VECTOR v; 
  v.M = N; 
  v.p = 2; 
  v.overflow = 0; 
  float * c; 
  unsigned long byteSize = (N*sizeof(unsigned long long)); 
  //Host 
  gpuErrchk(hipHostAlloc((void**)&v.vec,((v.M)*sizeof(unsigned long long)),hipHostMallocDefault)); 
  gpuErrchk(hipHostAlloc((void**)&c,((v.M)*sizeof(unsigned long long)),hipHostMallocDefault)); 

  randomInit(v.vec); 

  printf("Size of vec= %lu \n", byteSize); 

  printf("----------------Split up vector-------------------------\n"); 

  /*------------Basic Generic CUDA Setup------------------- */ 
  float * aC; 
   
  unsigned long long Nn = ceil(v.M / v.p); 
  unsigned long long bt = (long long)byteSize/v.p; 
  unsigned long long mem = (long long) (gMem-oneGB); 
  //printf("Nn=%llu, bt=%llu, mem=%llu",Nn,bt,mem); 
  while(bt>mem){ 
     
    v.p += 2; 
    bt = (long long)byteSize/v.p; 
    Nn = v.M/v.p;
    v.overflow = v.M%v.p; 
  } 

  hipEvent_t start,stop; 
  float time; 

  gpuErrchk(hipEventCreate(&start)); 
  gpuErrchk(hipEventCreate(&stop)); 
  //Timer START LETS GOOO! 
  gpuErrchk(hipEventRecord(start,0)); 
  //malloc 
  gpuErrchk(hipMalloc((void**)&aC, (Nn*sizeof( unsigned long long)))); 

  dim3 BLOCK(BLOCK_SIZE); 
  dim3 GRID((Nn+BLOCK.x-1/BLOCK.x)); 

 // printf("GRID(%d,%d,%d), BLOCK(%d,%d,%d)\n",GRID.x,GRID.y,GRID.z,BLOCK.x,BLOCK.y,BLOCK.z); 

//----------------------START LOOP--------------------------------; 

for (unsigned long long i = 0; i < v.M; i+=Nn){ 

    gpuErrchk(hipMemcpy(aC,v.vec+i,(Nn*sizeof(unsigned long long)),hipMemcpyHostToDevice)); 

    Incr<<<GRID,BLOCK,0>>>(aC,Nn,i); 

    gpuErrchk(hipMemcpy(c+i,aC,(Nn*sizeof(unsigned long long)),hipMemcpyDeviceToHost)); //i = N; 
  }   
  if (v.overflow) 
  { 
    gpuErrchk(hipMemcpy(aC,v.vec+(v.M-v.overflow),(v.overflow*sizeof(unsigned long long)),hipMemcpyHostToDevice)); 
    Incr<<<GRID,BLOCK,0>>>(aC,v.overflow,v.overflow); 
    gpuErrchk(hipMemcpy(c+(v.M-v.overflow),aC,(v.overflow*sizeof(unsigned long long)),hipMemcpyDeviceToHost)); 
  } 

//----------------------END LOOP-------------------------------- 
    hipDeviceSynchronize(); 
    gpuErrchk(hipEventRecord(stop,0)); 
    gpuErrchk(hipEventSynchronize(stop)); 
    gpuErrchk(hipEventElapsedTime(&time, start, stop)); 

    printf("Time Taken: %3.1f ms/n \n",time); 
    gpuErrchk(hipEventDestroy(start)); 
    gpuErrchk(hipEventDestroy(stop)); 
    printf("No stream\n"); 

  printf("\n freeing all vectors from memory\n"); 
  //  CheckI(v.vec,v.M,c); 
 
  gpuErrchk( hipHostFree( v.vec ) ); 
  gpuErrchk( hipFree( aC ) ); 
 
    gpuErrchk(hipEventRecord(stop1,0));  
    gpuErrchk(hipEventSynchronize(stop1));  
    gpuErrchk(hipEventElapsedTime(&time1, start1, stop1));  
  
    printf("Whole Time Taken: %6f s/n \n",time1/1000);  
    gpuErrchk(hipEventDestroy(start1));  
    gpuErrchk(hipEventDestroy(stop1));  
 
  return 0; 
} 
//Changed kernel to *3.3 
int CheckI(float * vv, unsigned long s, float *&c){ 
   
  for (int i = 0; i <=s ; ++i) 
  { 
    vv[i]+=1; 
 
    if (vv[i]!=c[i]) 
    { 
      printf("vv[%d]= %f, but c = %f\n",i,vv[i],c[i]); 
        return(0); 
    } 
  } 
  return (0); 
} 