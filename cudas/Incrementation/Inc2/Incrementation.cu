#include "hip/hip_runtime.h"
#include <time.h>       /* time */ 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include "ISplit.h" 
 
/////////////// MACROS and GLOBALS: ////////////// 
#define N 100000000 
#define BLOCK_SIZE 32 
#define oneGB 100000000 
long gMem; int gSize[3]; int wSize; int TPB;//max threads per block 
///////////////////////////// 
 
typedef struct{ 
   
  unsigned long long M;//size 
  unsigned int p; //partition 
  unsigned int overflow; //overflow 
  float * vec; 
}VECTOR; 
 
////////////////////////////////////////////////////////// 
int CheckI(float *vv, unsigned long s, float *&c); 
 
void randomInit(float* &data) 
{ 
  #pragma unroll 
    for (int i = 0; i <= N; i++){ 
        data[i] = rand()% (1000 + 1 - 1) + 1; 
        } 
} 
////////////////////////////////////////////////////////// 

/////////////////////////////////////////////////////////////////////////////////////// 
int main(int argc, char** argv){ 

  //Setup Check// 
  int Dev = 0; 
  hipDeviceProp_t pp; 
  setProp(Dev); 
  pp = getProp(); 
   
  hipEvent_t start1,stop1;   
  float time1;  
  gpuErrchk(hipEventCreate(&start1));  
  gpuErrchk(hipEventCreate(&stop1));
  gpuErrchk(hipEventRecord(start1,0));  
  hipEvent_t startP,stopP; 
  float timeP; 
  gpuErrchk(hipEventCreate(&startP)); 
  gpuErrchk(hipEventCreate(&stopP)); 
  gpuErrchk(hipEventRecord(startP,0)); 

  gMem = pp.totalGlobalMem; 
  gSize[0] = pp.maxGridSize[0]; gSize[1] = pp.maxGridSize[1]; gSize[2] = pp.maxGridSize[2]; 
  wSize = pp.warpSize; 
  TPB = pp.maxThreadsPerBlock; 
//  printf("total Global mem: %ld\n", gMem); 
//  printf("maxGridSize= %d,%d,%d \n",gSize[0],gSize[1],gSize[2]); 
//  printf("Warp Size: %d\n", wSize); 
//  printf(" TPB: %d\n", TPB); 
//----------------------------------------------------------- 
  srand(356); 

  printf("Initialised\n"); 

  VECTOR v; 
  v.M = N; 
  v.p =2; 
  v.overflow = 0; 

  float * c; 
  unsigned long byteSize = (N*sizeof(unsigned long long)); 
  //Host 
  gpuErrchk(hipHostAlloc((void**)&v.vec,((v.M)*sizeof(unsigned long long)),hipHostMallocDefault)); 
  gpuErrchk(hipHostAlloc((void**)&c,((v.M)*sizeof(unsigned long long)),hipHostMallocDefault)); 

  randomInit(v.vec); 

  printf("Size of vec= %lu \n", byteSize); 

  printf("----------------Split up vector-------------------------\n"); 

  /*------------Basic Generic CUDA Setup------------------- */ 

  unsigned long long Nn = ceil(v.M / v.p); 
  unsigned long long bt = (long long)byteSize/v.p; 
  unsigned long long mem = (long long) (gMem-oneGB); 
  //printf("Nn=%llu, bt=%llu, mem=%llu",Nn,bt,mem); 
   
  while((bt)>mem){ 
     
    v.p += 2; 
    bt = (long long)byteSize/v.p; 
    Nn = v.M/v.p; 
    v.overflow = v.M%v.p; 
  } 

  dim3 BLOCK(BLOCK_SIZE); 
  dim3 GRID(Nn+BLOCK.x-1/BLOCK.x); 

  //printf("GRID(%lu,%d,%d), BLOCK(%d,%d,%d)\n",GRID.x,GRID.y,GRID.z,BLOCK.x,BLOCK.y,BLOCK.z); 
  //printf("partition = %lu\n",v.p); 

  hipStream_t stream0; 
  hipEvent_t start,stop; 
  float time; 

  gpuErrchk(hipEventCreate(&start)); 
  gpuErrchk(hipEventCreate(&stop)); 
  gpuErrchk( hipStreamCreate( &stream0)); 
  //Timer START LETS GOOO! 
  gpuErrchk(hipEventRecord(start,0)); 
  //malloc 
  float * aC; 
  gpuErrchk(hipMalloc((void**)&aC, (Nn*sizeof( unsigned long long)))); 

//----------------------START LOOP--------------------------------// 

for (unsigned long long i = 0; i <= v.M-v.overflow; i+=Nn){ 

    gpuErrchk(hipMemcpyAsync(aC,v.vec+i,(Nn*sizeof(unsigned long long)),hipMemcpyHostToDevice,stream0)); 

    Incr<<<GRID,BLOCK,0,stream0>>>(aC,Nn,i); 

    gpuErrchk(hipMemcpyAsync(c+i,aC,(Nn*sizeof(unsigned long long)),hipMemcpyDeviceToHost,stream0)); //i = N; 
  } 
  if (v.overflow) 
  { 
    gpuErrchk(hipMemcpyAsync(aC,v.vec+(v.M-v.overflow),(v.overflow*sizeof(unsigned long long)),hipMemcpyHostToDevice,stream0)); 
    Incr<<<GRID,BLOCK,0,stream0>>>(aC,v.overflow,v.overflow); 
    gpuErrchk(hipMemcpyAsync(c+(v.M-v.overflow),aC,(v.overflow*sizeof(unsigned long long)),hipMemcpyDeviceToHost,stream0)); 
  } 
//----------------------END LOOP--------------------------------// 

    gpuErrchk(hipStreamSynchronize(stream0)); // Tell CPU to hold his horses and wait 
    hipDeviceSynchronize(); 
    gpuErrchk(hipEventRecord(stop,0)); 
    gpuErrchk(hipEventSynchronize(stop)); 
    gpuErrchk(hipEventElapsedTime(&time, start, stop)); 
    printf("Time Taken: %3.1f ms \n",time); 
    gpuErrchk(hipStreamDestroy(stream0)); 
    gpuErrchk(hipEventDestroy(start)); 
    gpuErrchk(hipEventDestroy(stop)); 
    printf("1 Stream\n"); 
 
    printf("\n freeing all vectors from memory\n"); 
 
    //CheckI(v.vec,v.M,c); 
 
    gpuErrchk( hipHostFree( v.vec ) ); 
    gpuErrchk( hipFree( aC ) ); 
  gpuErrchk(hipEventRecord(stopP,0)); 
  gpuErrchk(hipEventSynchronize(stopP)); 
  gpuErrchk(hipEventElapsedTime(&timeP, startP, stopP)); 
  gpuErrchk(hipEventDestroy(startP)); 
  gpuErrchk(hipEventDestroy(stopP)); 
 
    gpuErrchk(hipEventRecord(stop1,0));   
    gpuErrchk(hipEventSynchronize(stop1));  
    gpuErrchk(hipEventElapsedTime(&time1, start1, stop1));  
    gpuErrchk(hipEventDestroy(stop1));    
    gpuErrchk(hipEventDestroy(start1));  
    printf("Parallel Time Taken: %3.1f ms \n",time);  
    time1 +=time;  
    printf("Full Time Taken: %6f seconds \n",time1/1000.0000);  
  return 0; 
} 
 
int CheckI(float * vv, unsigned long s, float *&c){ 
   
  for (int i = 0; i <=s ; ++i) 
  { 
    vv[i]+=1; 
 
    if (vv[i]!=c[i]) 
    { 
      printf("vv[%d]= %f, but c = %f\n",i,vv[i],c[i]); 
        return(0); 
    } 
  } 
  return (42); 
} 