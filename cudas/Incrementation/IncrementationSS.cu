#include "hip/hip_runtime.h"
#include <stdio.h>      /* printf, NULL */ 
#include <stdlib.h>     /* srand, rand */ 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include "ISplit.h" 
 
/////////////// MACROS and GLOBALS: ////////////// 
#define N 400000000 
#define BLOCK_SIZE 32 
#define oneGB 100000000 
long gMem; int gSize[3]; int wSize; int TPB;//max threads per block 
///////////////////////////// 
 
typedef struct{ 
   
  unsigned long long M;//size 
  unsigned int p; //partition 
  unsigned int overflow; //overflow 
  float * vec; 
}VECTOR; 
 
////////////////////////////////////////////////////////// 
int CheckI(float *vv, unsigned long s, float *&c); 
 
void randomInit(float* &data) 
{ 
  #pragma unroll 
    for (int i = 0; i <= N; i++){ 
        data[i] = rand()% (1000 + 1 - 1) + 1; 
        } 
} 
////////////////////////////////////////////////////////// 
 
/////////////////////////////////////////////////////////////////////////////////////// 
int main(int argc, char** argv){ 
 
  //Setup Check// 
  int Dev = 0; 
  hipDeviceProp_t pp; 
  setProp(Dev); 
  pp = getProp(); 
   
  hipEvent_t start1,stop1;  
  float time1; 
  gpuErrchk(hipEventCreate(&start1)); 
  gpuErrchk(hipEventCreate(&stop1)); 
  gpuErrchk(hipEventRecord(start1,0)); 
 
  gMem = pp.totalGlobalMem; 
  gSize[0] = pp.maxGridSize[0]; gSize[1] = pp.maxGridSize[1]; gSize[2] = pp.maxGridSize[2]; 
  wSize = pp.warpSize; 
  TPB = pp.maxThreadsPerBlock; 
//  printf("total Global mem: %ld\n", gMem); 
//  printf("maxGridSize= %d,%d,%d \n",gSize[0],gSize[1],gSize[2]); 
//  printf("Warp Size: %d\n", wSize); 
//  printf(" TPB: %d\n", TPB); 
//----------------------------------------------------------- 
  srand(356); 
 
  printf("Initialised\n"); 
//  printf("Creating Template vector\n"); 
 
  VECTOR v; 
  v.M = N; 
  v.p =2; 
  v.overflow = 0; 
 
  float * c; 
  unsigned long byteSize = (N*sizeof(long long)); 
  //Host 
  gpuErrchk(hipHostAlloc((void**)&v.vec,((v.M)*sizeof(long long)),hipHostMallocDefault)); 
  gpuErrchk(hipHostAlloc((void**)&c,((v.M)*sizeof(long long)),hipHostMallocDefault)); 
 
  randomInit(v.vec); 
 
 printf("Size of vec= %lu \n", byteSize); 
 
  /*------------Basic Generic CUDA Setup------------------- */ 
 
  unsigned long Nn = ceil(v.M / v.p); 
  unsigned long bt = (long long)byteSize/v.p; 
  unsigned long long mem = (long long) (gMem-oneGB); 
   
  while((bt*2)>mem){ 
     
    v.p += 2; 
    bt = (long long)byteSize/v.p; 
    Nn = v.M/v.p; 
    v.overflow = v.M%v.p; 
  } 

  dim3 BLOCK(BLOCK_SIZE); 
   dim3 GRID(Nn+BLOCK.x-1/BLOCK.x); 
 
  //printf("GRID(%lu,%d,%d), BLOCK(%d,%d,%d)\n",GRID.x,GRID.y,GRID.z,BLOCK.x,BLOCK.y,BLOCK.z); 
  //printf("partition = %lu\n",v.p); 
  //printf("overflow= %d \n",v.overflow); 
  hipStream_t stream0; 
  hipStream_t stream1; 
 
  hipEvent_t start,stop; 
  float time; 
 
  gpuErrchk(hipEventCreate(&start)); 
  gpuErrchk(hipEventCreate(&stop)); 
  gpuErrchk( hipStreamCreate( &stream0)); 
  gpuErrchk( hipStreamCreate( &stream1)); 
  //Timer START LETS GOOO! 
  gpuErrchk(hipEventRecord(start,0)); 
  //malloc 
  float * aC; 
  float * aC1; 
  gpuErrchk(hipMalloc((void**)&aC, (Nn*sizeof(long long)))); 
  gpuErrchk(hipMalloc((void**)&aC1, (Nn*sizeof(long long)))); 
 
//----------------------START LOOP--------------------------------); 
 
for (unsigned long long i = 0; i < v.M-v.overflow; i+=Nn*2){ //Nn*2 because 2 streams 
 
    gpuErrchk(hipMemcpyAsync(aC,v.vec+i,(Nn*sizeof(long long)),hipMemcpyHostToDevice,stream0)); 
    gpuErrchk(hipMemcpyAsync(aC1,v.vec+(i+Nn),(Nn*sizeof(long long)),hipMemcpyHostToDevice,stream1));  
     
    Incr<<<GRID,BLOCK,0,stream0>>>(aC,Nn,i); 
    Incr<<<GRID,BLOCK,0,stream1>>>(aC1,Nn,i); 
     
    gpuErrchk(hipMemcpyAsync(c+i,aC,(Nn*sizeof(long long)),hipMemcpyDeviceToHost,stream0)); //i = N; 
    gpuErrchk(hipMemcpyAsync(c+(i+Nn),aC1,(Nn*sizeof(long long)),hipMemcpyDeviceToHost,stream1)); 
  } 
   
    if (v.overflow) 
    { 
      gpuErrchk(hipMemcpyAsync(aC,v.vec+(v.M-v.overflow),(v.overflow*sizeof(long long)),hipMemcpyHostToDevice,stream1)); 
      Incr<<<GRID,BLOCK,0,stream1>>>(aC,v.overflow,v.overflow); 
      gpuErrchk(hipMemcpyAsync(c+(v.M-v.overflow),aC,(v.overflow*sizeof(long long)),hipMemcpyDeviceToHost,stream1)); 
    } 

    gpuErrchk(hipStreamSynchronize(stream0)); // Tell CPU to hold his horses and wait 
    gpuErrchk(hipStreamSynchronize(stream1)); // Tell CPU to hold his horses and wait 
    hipDeviceSynchronize(); 
    gpuErrchk(hipEventRecord(stop,0)); 
    gpuErrchk(hipEventSynchronize(stop)); 
    gpuErrchk(hipEventElapsedTime(&time, start, stop)); 
 
    gpuErrchk(hipStreamDestroy(stream0)); 
    gpuErrchk(hipStreamDestroy(stream1)); 
    gpuErrchk(hipEventDestroy(start)); 
    gpuErrchk(hipEventDestroy(stop)); 
    printf("2 Stream\n"); 
 
    //CheckI(v.vec,v.M,c); 
     
    gpuErrchk( hipHostFree( v.vec ) ); 
    gpuErrchk( hipFree( aC ) ); 
    gpuErrchk( hipFree( aC1 ) ); 
 
    gpuErrchk(hipEventRecord(stop1,0));  
    gpuErrchk(hipEventSynchronize(stop1)); 
    gpuErrchk(hipEventElapsedTime(&time1, start1, stop1)); 
    gpuErrchk(hipEventDestroy(stop1));   
    gpuErrchk(hipEventDestroy(start1)); 
    printf("Parallel Time Taken: %3.1f ms \n",time); 
    time1 +=time; 
    printf("Full Time Taken: %6f seconds \n",time1/1000.0000); 
return 0; 
} 
 
int CheckI(float * vv, unsigned long s, float *&c){ 
   
  for (int i = 0; i <=s ; ++i) 
  { 
    vv[i]*=3.3; 
 
    if (vv[i]!=c[i]) 
    { 
      printf("vv[%d]= %f, but c = %f\n",i,vv[i],c[i]); 
      return(42); 
    } 
  } 
  return (0); 
}