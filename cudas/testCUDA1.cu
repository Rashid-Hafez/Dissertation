#include<time.h>
#include<stdio.h>
#include<hip/hip_runtime.h>

#define N 30
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//Global function is able to be accessed by GPU and CPU
__global__ void vectorAdd(int *a,int *b,int *c){
        if(blockIdx.x<N){
                c[blockIdx.x] = a[blockIdx.x]+b[blockIdx.x];
        }
}
void randomInit(int* data, int size)
{
    for (int i = 0; i < size; ++i){
        data[i] = rand()% (1000 + 1 - 1) + 1;
        	if(i<5){
			printf("\n%d",data[i]);
		}
        }
}

int main(){

printf("hello\n");

int size = N *sizeof(int);

int *a, *b, *c; //host vectors
int *aC,*bC,*cC;//cuda vectors
struct timespec start,stop;

        printf("\n Code to add vectors A and B");
        hipMalloc((void**)&aC, size);
        hipMalloc((void**)&bC,size);
        hipMalloc((void**)&cC, size);

        a= (int *)malloc(size);
        b = (int *)malloc(size);
        c=(int *)malloc(size);
        randomInit(a,N); randomInit(b,N);

        gpuErrchk(hipMemcpy(aC,a,size,hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(bC,b,size,hipMemcpyHostToDevice));
	
	clock_gettime(CLOCK_REALTIME,&start);
	//Create kernel of N blocks holding 1 threads
        vectorAdd<<<N,1>>>(aC,bC,cC);//can do <<<N,1>>> for parralel
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk(hipDeviceSynchronize());
        
	gpuErrchk(hipMemcpy(c,cC,size,hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME,&stop);
	printf("\n printing 20 results of C");
                for(int i=0;i<20;i++){
                        printf("\n%d",c[i]);
                }

        printf("\n freeing all vectors from memory");
        free(a); free(b); free(c);
        hipFree(aC); hipFree(bC); hipFree(cC);//changed to cuda free
return 0;
}
